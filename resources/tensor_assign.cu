#include "hip/hip_runtime.h"
#include "index_system.h"
#include "datatypes.h"

using namespace think;
using namespace tensor::index_system;


template<typename dest_type,
	 typename src_type>
__device__
void assign(dest_type* dest, const general_index_system&  dest_idx_system,
	    const src_type* src, const general_index_system& src_idx_system,
	    int n_elems)
{
  int elem_idx = blockDim.x * blockIdx.x + threadIdx.x;
  if ( elem_idx < n_elems ) {
    dest[dest_idx_system(elem_idx)] = static_cast<dest_type>( src[src_idx_system(elem_idx)] );
  }
};



#define DATATYPE_2_ITERATOR(lhs_dtype, lhs_ext, rhs_dtype, rhs_ext)    \
extern "C"								\
__global__								\
void tensor_assign##lhs_ext##rhs_ext(					\
  datatype::lhs_dtype* dest, EXPLODE_IDX_SYSTEM(dest),			\
  const datatype::rhs_dtype* src, EXPLODE_IDX_SYSTEM(src),		\
  int n_elems)								\
{									\
  assign(dest, ENCAPSULATE_IDX_SYSTEM(dest),				\
	 src, ENCAPSULATE_IDX_SYSTEM(src),				\
	 n_elems);							\
}

ITERATE_2_DATATYPES;
#undef DATATYPE_2_ITERATOR
